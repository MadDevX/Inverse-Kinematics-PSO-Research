#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <thrust/extrema.h>
#include "Particle.h"
#include "utility_kernels.cuh"
#include "matrix_operations.cuh"
#include "quaternion_operations.cuh"
#include "vector_operations.cuh"
#include "ik_constants.h"

__constant__ float locality = -0.1f;
__constant__ float angleWeight = 0.05f;
__constant__ float errorThreshold = 0.1f;

__device__ float calculateDistance(KinematicChainCuda chain, Particle particle, float3 targetPosition)
{
	Matrix model = createMatrix(1.0f);
	model = translateMatrix(model, chain._shoulderPosition);
	model = rotateEuler(model, particle.positions.shoulderRotX, particle.positions.shoulderRotY, particle.positions.shoulderRotZ);
	model = translateMatrix(model, make_float3(chain._armLength, 0.0f, 0.0f));
	model = rotateEuler(model, particle.positions.elbowRotX, particle.positions.elbowRotY, particle.positions.elbowRotZ);
	model = translateMatrix(model, make_float3(chain._forearmLength, 0.0f, 0.0f));
	float4 position = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	position = multiplyMatByVec(model, position);
	float3 diff = make_float3(position.x - targetPosition.x, position.y - targetPosition.y, position.z - targetPosition.z);
	float3 diffShoulder = make_float3(chain._shoulderRotation.x - particle.positions.shoulderRotX, chain._shoulderRotation.y - particle.positions.shoulderRotY, chain._shoulderRotation.z - particle.positions.shoulderRotZ);
	float3 diffElbow = make_float3(chain._elbowRotation.x - particle.positions.elbowRotX, chain._elbowRotation.y - particle.positions.elbowRotY, chain._elbowRotation.z - particle.positions.elbowRotZ);
	float distance = magnitudeSqr(diff);
	return distance + angleWeight * (magnitudeSqr(diffShoulder) + magnitudeSqr(diffElbow));
}

__device__ Matrix calculateModelMatrix(NodeCUDA *chain, int nodeIndex)
{
	if (nodeIndex == 0)
	{	
		Matrix matrix = createMatrix(1.0f);
		matrix = translateMatrix(matrix, chain[nodeIndex].position);
		matrix = rotateMatrix(matrix, chain[nodeIndex].rotation);
		return matrix;
	}
	else
	{
		Matrix matrix = calculateModelMatrix(chain, chain[nodeIndex].parentIndex);
		matrix = rotateMatrix(matrix, chain[nodeIndex].rotation);
		matrix = translateMatrix(matrix, make_float3(chain[nodeIndex].length,0.0f,0.0f));
		return matrix;
	}
}

__device__ float calculateDistanceNew(NodeCUDA *chain, ParticleNew particle)
{
	float quaternionDifference = 0.0f;
	float distance = 0.0f;

	for(int ind = 1; ind <= DEGREES_OF_FREEDOM / 3; ind++)
	{
		float4 chainQuaternion = chain[ind].rotation;
		float4 particleQuaternionRotation = eulerToQuaternion(make_float3(particle.positions[(ind - 1) * 3],
			particle.positions[(ind - 1) * 3 + 1],
			particle.positions[(ind - 1) * 3 + 2]));

		quaternionDifference = quaternionDifference + magnitudeSqr(chainQuaternion - particleQuaternionRotation);
		
		if (chain[ind].nodeType == NodeType::effectorNode)
		{		
			Matrix model = calculateModelMatrix(chain, ind);
			float4 position = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
			position = multiplyMatByVec(model, position);

			distance = distance + magnitudeSqr(make_float3(position.x - chain[ind].targetPosition.x,
												  position.y - chain[ind].targetPosition.y,
												  position.z - chain[ind].targetPosition.z));
		}
		
	}

	return distance + angleWeight * (quaternionDifference);
}

__global__ void simulateParticlesNewKernel(ParticleNew *particles, float *bests, hiprandState_t *randoms, int size, NodeCUDA *chain, Config config, CoordinatesNew global, float globalMin)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for (int i = id; i < size; i += stride)
	{

		
		for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg++)
		{
			particles[i].velocities[deg] = config._inertia * particles[i].velocities[deg] +
				config._local * hiprand_uniform(&randoms[i]) * (particles[i].localBest[deg] - particles[i].positions[deg]) +
				config._global * hiprand_uniform(&randoms[i]) * (global.positions[deg]- particles[i].positions[deg]);

			particles[i].positions[deg] += particles[i].velocities[deg];

			
		}

		for (int ind = 1; ind <= DEGREES_OF_FREEDOM/3; ind++)
		{
			int deg = (ind - 1) * 3;
			particles[i].positions[deg]   =   clamp(particles[i].positions[deg], chain[ind].minRotation.x, chain[ind].maxRotation.x);
			particles[i].positions[deg + 1] = clamp(particles[i].positions[deg+1], chain[ind].minRotation.y, chain[ind].maxRotation.y);
			particles[i].positions[deg + 2] = clamp(particles[i].positions[deg+2], chain[ind].minRotation.z, chain[ind].maxRotation.z);
		}	
		float currentDistance = calculateDistanceNew(chain, particles[i]);
		
		if (currentDistance < bests[i])
		{
			
			bests[i] = currentDistance;
			for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg++)
			{
				particles[i].localBest[deg] = particles[i].positions[deg];
				
			}
			
		}
	}
}


__global__ void initParticlesNewKernel(ParticleNew *particles, float *localBests, hiprandState_t *randoms, NodeCUDA * chain, int size)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for (int i = id; i < size; i += stride)
	{
		if (hiprand_uniform(&randoms[i]) > locality)
		{

			for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg+=3)
			{

				float3 euler = quaternionToEuler(chain[(deg/3)+1].rotation);
				particles[i].positions[deg] = euler.x;
				particles[i].positions[deg+1] = euler.y;
				particles[i].positions[deg+2] = euler.z;		
			}
			
		}
		else
		{
			
			for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg += 3)
			{
				//Uniform distribution of particles across the domain
				int chainIndex = (deg / 3) + 1;
				float3 eulerMaxConstraint = quaternionToEuler(chain[chainIndex].maxRotation);
				float3 eulerMinConstraint = quaternionToEuler(chain[chainIndex].minRotation);
				
				particles[i].positions[deg] = (hiprand_uniform(&randoms[i]) * (eulerMaxConstraint.x - eulerMinConstraint.x)) + eulerMinConstraint.x;
				particles[i].positions[deg+1] = (hiprand_uniform(&randoms[i]) * (eulerMaxConstraint.y - eulerMinConstraint.y)) + eulerMinConstraint.y;
				particles[i].positions[deg+2] = (hiprand_uniform(&randoms[i]) * (eulerMaxConstraint.z - eulerMinConstraint.z)) + eulerMinConstraint.z;
			}
		}

		//Init bests with current data
		for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg += 1)
		{
			particles[i].velocities[deg] = hiprand_uniform(&randoms[i]) * 2.0f - 1.0f;
			particles[i].localBest[deg] = particles[i].positions[deg];
		}

		//Calculate bests
		localBests[i] = calculateDistanceNew(chain, particles[i]);
	}

}



hipError_t calculatePSONew(ParticleNew *particles, float *bests, hiprandState_t *randoms, int size, NodeCUDA *chain, Config config, CoordinatesNew *result)
{
	hipError_t status;
	CoordinatesNew global;
	float globalMin;
	int numBlocks = (size + blockSize - 1) / blockSize;

	initParticlesNewKernel << <numBlocks, blockSize >> > (particles, bests, randoms, chain, size);
	checkCuda(status = hipGetLastError());
	if (status != hipSuccess) return status;
	checkCuda(status = hipDeviceSynchronize());


	float *globalBest = thrust::min_element(thrust::host, bests, bests + size);
	int globalIndex = globalBest - bests;
	printf("global index = %d \n", globalIndex);
	for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg++)
	{
		global.positions[deg] = particles[globalIndex].localBest[deg];
	}
	
	globalMin = bests[globalIndex];

	for (int i = 0; i < config._iterations; i++)
	{
		simulateParticlesNewKernel << <numBlocks, blockSize >> > (particles, bests, randoms, size, chain, config, global, globalMin);
		checkCuda(status = hipGetLastError());
		if (status != hipSuccess) return status;
		checkCuda(status = hipDeviceSynchronize());
		globalBest = thrust::min_element(thrust::host, bests, bests + size);
		globalIndex = globalBest - bests;
		for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg++)
		{
			global.positions[deg] = particles[globalIndex].localBest[deg];
		}

		globalMin = bests[globalIndex];
	}

	*result = global;

	return status;
}