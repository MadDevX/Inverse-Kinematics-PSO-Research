#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <thrust/extrema.h>
#include "Particle.h"
#include "utility_kernels.cuh"
#include "matrix_operations.cuh"
#include "quaternion_operations.cuh"
#include "vector_operations.cuh"
#include "ik_constants.h"


__constant__ float angleWeight = 3.0f;
__constant__ float errorThreshold = 0.1f;



__device__ Matrix calculateModelMatrix(NodeCUDA *chain, ParticleNew *particle, int nodeIndex)
{
	Matrix matrix = createMatrix(1.0f);
	while (nodeIndex != 0)
	{
		int particleIndex = (nodeIndex - 1) * 3;
		float3 particleEulerRotation = make_float3(particle->positions[particleIndex],
			particle->positions[particleIndex + 1],
			particle->positions[particleIndex + 2]);

		Matrix tempMat = createMatrix(1.0f);
		tempMat = rotateEuler(tempMat, particleEulerRotation);
		tempMat = translateMatrix(tempMat, make_float3(chain[nodeIndex].length, 0.0f, 0.0f));
		matrix = multiplyMatrices(tempMat,matrix);
		nodeIndex = chain[nodeIndex].parentIndex;
	}
	Matrix originMatrix = createMatrix(1.0f);
	originMatrix = translateMatrix(originMatrix, chain[nodeIndex].position);
	originMatrix = rotateEuler(originMatrix, chain[nodeIndex].rotation);
	return multiplyMatrices(originMatrix, matrix);
}


//Ewentualnie kolizje moga byc sprawdzane dla odcinka i colliderow, wtedy przekazujemy do funkcji 2 x float3 i liste colliderow.
//Wtedy checkCollision byloby wywolane w wewnetrznej petli calculateDistance.
__device__ bool checkCollisions(NodeCUDA *chain, ParticleNew particle /*, Collider* colliders*/)
{
	return false;
}

__device__ float calculateDistanceNew(NodeCUDA *chain, ParticleNew particle)
{
	float rotationDifference = 0.0f;
	float distance = 0.0f;

	for(int ind = 1; ind <= DEGREES_OF_FREEDOM / 3; ind++)
	{
		float3 chainRotation = chain[ind].rotation;
		float3 particleRotation = make_float3(
			particle.positions[(ind - 1) * 3],
			particle.positions[(ind - 1) * 3 + 1],
			particle.positions[(ind - 1) * 3 + 2]);

		rotationDifference = rotationDifference + magnitudeSqr(chainRotation - particleRotation);
		
		if (chain[ind].nodeType == NodeType::effectorNode)
		{		
			Matrix model = calculateModelMatrix(chain,&particle, ind);
			float4 position = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
			position = multiplyMatByVec(model, position);

			float distTmp = magnitudeSqr(make_float3(
													position.x - chain[ind].targetPosition.x,
													position.y - chain[ind].targetPosition.y,
													position.z - chain[ind].targetPosition.z));
		
			distance = distance + distTmp;
		}
		
		
	}

	return distance + angleWeight/(DEGREES_OF_FREEDOM / 3) * rotationDifference;
}

__global__ void simulateParticlesNewKernel(ParticleNew *particles, float *bests, hiprandState_t *randoms, int size, NodeCUDA *chain, Config config, CoordinatesNew global, float globalMin)
{
	extern __shared__ NodeCUDA sharedChain[];

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for (int i = id % blockDim.x; i < (DEGREES_OF_FREEDOM / 3) + 1; i++)
	{
		sharedChain[i] = chain[i];
	}

	for (int i = id; i < size; i += stride)
	{
		
		for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg++)
		{
			particles[i].velocities[deg] = config._inertia * hiprand_uniform(&randoms[i]) * particles[i].velocities[deg] +
										   config._local   * hiprand_uniform(&randoms[i]) * (particles[i].localBest[deg] - particles[i].positions[deg]) +
										   config._global  * hiprand_uniform(&randoms[i]) * (global.positions[deg]- particles[i].positions[deg]);

			particles[i].positions[deg] += particles[i].velocities[deg];

			
		}

		//for (int ind = 1; ind <= DEGREES_OF_FREEDOM/3; ind++)
		//{
		//	int deg = (ind - 1) * 3;
		//	particles[i].positions[deg]   =   clamp(particles[i].positions[deg], chain[ind].minRotation.x, chain[ind].maxRotation.x);
		//	particles[i].positions[deg + 1] = clamp(particles[i].positions[deg+1], chain[ind].minRotation.y, chain[ind].maxRotation.y);
		//	particles[i].positions[deg + 2] = clamp(particles[i].positions[deg+2], chain[ind].minRotation.z, chain[ind].maxRotation.z);
		//}	
		float currentDistance = calculateDistanceNew(sharedChain, particles[i]);
		
		if (currentDistance < bests[i])
		{
			
			bests[i] = currentDistance;
			for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg++)
			{
				particles[i].localBest[deg] = particles[i].positions[deg];
			}
			
		}
	}
}


__global__ void initParticlesNewKernel(ParticleNew *particles, float *localBests, hiprandState_t *randoms, NodeCUDA * chain, int size)
{
	extern __shared__ NodeCUDA sharedChain[];

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for (int i = id % blockDim.x; i < (DEGREES_OF_FREEDOM / 3) + 1; i++)
	{
		sharedChain[i] = chain[i];
	}


	
	for (int i = id; i < size; i += stride)
	{
	
		for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg += 3)
		{
			//Uniform distribution of particles across the domain
			int chainIndex = (deg / 3) + 1;
			float3 eulerMaxConstraint = sharedChain[chainIndex].maxRotation;
			float3 eulerMinConstraint = sharedChain[chainIndex].minRotation;

			//printf("maxconstraint x %f\n", chain[chainIndex].maxRotation.x);
			//printf("maxconstraint y %f\n", chain[chainIndex].maxRotation.y);
			//printf("maxconstraint z %f\n", chain[chainIndex].maxRotation.z);


			//printf("quaterniondiff - deg %d : %f\n",deg, eulerMaxConstraint.z - eulerMinConstraint.z);
			//printf("quaterniondiff - deg %d : %f\n",deg+1, eulerMaxConstraint.x - eulerMinConstraint.x);
			//printf("quaterniondiff - deg %d : %f\n",deg+2, eulerMaxConstraint.y - eulerMinConstraint.y);
			//particles[i].positions[deg] =     (hiprand_uniform(&randoms[i])    *6.28f - 3.14f); //(hiprand_uniform(&randoms[i]) * (eulerMaxConstraint.x - eulerMinConstraint.x)) + eulerMinConstraint.x;
			//particles[i].positions[deg + 1] = (hiprand_uniform(&randoms[i])*6.28f - 3.14f);// (hiprand_uniform(&randoms[i]) * (eulerMaxConstraint.y - eulerMinConstraint.y)) + eulerMinConstraint.y;
			//particles[i].positions[deg + 2] = (hiprand_uniform(&randoms[i])*6.28f - 3.14f);// (hiprand_uniform(&randoms[i]) * (eulerMaxConstraint.z - eulerMinConstraint.z)) + eulerMinConstraint.z;
			float3 eulerRot = sharedChain[chainIndex].rotation;
			particles[i].positions[deg] = eulerRot.x;
			particles[i].positions[deg + 1] = eulerRot.y;
			particles[i].positions[deg + 2] = eulerRot.z;

		}

		//Init bests with current data
		for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg += 1)
		{
			particles[i].velocities[deg] = hiprand_uniform(&randoms[i]) * 2.0f - 1.0f;
			particles[i].localBest[deg] = particles[i].positions[deg];
		}

		//Calculate bests
		localBests[i] = calculateDistanceNew(sharedChain, particles[i]);
		
	}

}



hipError_t calculatePSONew(ParticleNew *particles, float *bests, hiprandState_t *randoms, int size, NodeCUDA *chain, Config config, CoordinatesNew *result)
{
	hipError_t status;
	CoordinatesNew global;
	float globalMin;
	int numBlocks = (size + blockSize - 1) / blockSize;
	int sharedMemorySize = sizeof(NodeCUDA)*((DEGREES_OF_FREEDOM / 3) + 1);

	initParticlesNewKernel << <numBlocks, blockSize, sharedMemorySize>> > (particles, bests, randoms, chain, size);
	checkCuda(status = hipGetLastError());
	if (status != hipSuccess) return status;
	checkCuda(status = hipDeviceSynchronize());

	float *globalBest = thrust::min_element(thrust::host, bests, bests + size);

	int globalIndex = globalBest - bests;

	for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg++)
	{
		global.positions[deg] = particles[globalIndex].localBest[deg];
	}
	
	globalMin = bests[globalIndex];

	for (int i = 0; i < config._iterations; i++)
	{
		simulateParticlesNewKernel << <numBlocks, blockSize, sharedMemorySize >> > (particles, bests, randoms, size, chain, config, global, globalMin);
		checkCuda(status = hipGetLastError());
		if (status != hipSuccess) return status;
		checkCuda(status = hipDeviceSynchronize());
		globalBest = thrust::min_element(thrust::host, bests, bests + size);
		globalIndex = globalBest - bests;
		if (globalMin > bests[globalIndex])
		{
			globalMin = bests[globalIndex];
			for (int deg = 0; deg < DEGREES_OF_FREEDOM; deg++)
			{
				global.positions[deg] = particles[globalIndex].localBest[deg];
			}
		}
		for (int i = 0; i < size; i++)
		{
			//printf("\tODLGLOSC %d - %f: \n",i, bests[i]);
		}
		

	}

	*result = global;
	//printf("Global Min: %f; Index = %d\n", globalMin, globalIndex);
	return status;
}